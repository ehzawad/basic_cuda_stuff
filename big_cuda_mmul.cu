#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#define N 1000

__global__ void multiply(int* A, int* B, int* C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int sum = 0;
  for (int k = 0; k < N; k++) {
    sum += A[row * N + k] * B[k * N + col];
  }

  C[row * N + col] = sum;
}

int main() {
  int *A, *B, *C;
  hipMallocManaged(&A, N * N * sizeof(int));
  hipMallocManaged(&B, N * N * sizeof(int));
  hipMallocManaged(&C, N * N * sizeof(int));

  for (int i = 0; i < N * N; i++) {
    A[i] = 2;
    B[i] = 3;
  }

  auto start = std::chrono::high_resolution_clock::now();

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  multiply<<<numBlocks, threadsPerBlock>>>(A, B, C);
  hipDeviceSynchronize();

  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

  std::cout << "Time taken by GPU: " << duration.count() << " microseconds" << std::endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);

  return 0;
}
