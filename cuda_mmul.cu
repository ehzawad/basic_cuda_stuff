
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <chrono>

#define X 3
#define Y 6
#define Z 10

__global__ void multiplyMatrices(int *a, int *b, int *c) {
  int x = blockIdx.x;
  int y = blockIdx.y;
  int z = threadIdx.x;

  c[x * Y * Z + y * Z + z] = a[x * Y * Z + y * Z + z] * b[x * Y * Z + y * Z + z];
}

int main() {
  int a[X * Y * Z];
  int b[X * Y * Z];
  int c[X * Y * Z];

  for(int i = 0; i < X * Y * Z; i++) {
    a[i] = i + 1;
    b[i] = i + 1;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, X * Y * Z * sizeof(int));
  hipMalloc(&d_b, X * Y * Z * sizeof(int));
  hipMalloc(&d_c, X * Y * Z * sizeof(int));

  hipMemcpy(d_a, a, X * Y * Z * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, X * Y * Z * sizeof(int), hipMemcpyHostToDevice);

  dim3 blocks(X, Y);
  dim3 threads(Z);

  auto start = std::chrono::high_resolution_clock::now();
  multiplyMatrices<<<blocks, threads>>>(d_a, d_b, d_c);
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();

  hipMemcpy(c, d_c, X * Y * Z * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // Print result
  printf("Result (3D Matrix):\n");
  for (int x = 0; x < X; x++) {
    for (int y = 0; y < Y; y++) {
      for (int z = 0; z < Z; z++) {
        printf("%d ", c[x * Y * Z + y * Z + z]);
      }
      printf("\n");
    }
    printf("\n");
  }

  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Execution Time (CUDA): " << duration.count() << " microseconds\n";
}

